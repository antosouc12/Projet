#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>



__global__ void MatrixMul(float *M1,float *M2,float *M3, int n, int m, int p){

    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx<n*p){ 
        int x=idx/p;
        int y=idx%p;
        float tmp;
        for (int i=0; i<m;i++){
            tmp+=M1[x*m+i]*M2[i*p+y];
        }
        M3[idx]=tmp;
    }

}
