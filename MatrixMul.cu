#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>



// __device__ float Mulsub(){
//     float out;

// }

__global__ void MatrixMul(float *M1,float *M2,float *M3, int n, int m, int p){

    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx<n*p){ 
        int x=idx/p;
        int y=idx%p;
        float tmp;
        for (int i=0; i<m;i++){
            tmp+=M1[x*m+i]*M2[i*p+y];
        }
        M3[idx]=tmp;
    }

}

// __host__ int main(void){


//     int n=3;
//     int p=3;

//     float *M_h1, *M_h2,*M_h3, *M_d1,*M_d2,*M_d3;
//     printf("Etape 1 \n");
//     M_h1=(float*)malloc(n*p*sizeof(float));
//     M_h2=(float*)malloc(n*p*sizeof(float));
//     M_h3=(float*)malloc(n*p*sizeof(float));
//     // for (int m=0;m<n;m++){
//     //     M_h1[m]=(float*)malloc(p*sizeof(float));
//     //     M_h2[m]=(float*)malloc(p*sizeof(float));
//     //     M_h3[m]=(float*)malloc(p*sizeof(float));
//     // }

//     int Nthread=p;
//     int Nblock= n;
    
//     for (int i=0;i<n*p;i++){
        
//         M_h1[i]=1;
//         M_h2[i]=2;
        
//     }
//     printf("Etape 2 \n");
//     cudaMalloc((void **) &M_d1, n*p*sizeof(float));
//     cudaMalloc((void **) &M_d2, n*p*sizeof(float));
//     cudaMalloc((void **) &M_d3, n*p*sizeof(float));

//     cudaMemcpy(M_d1,M_h1,n*p*sizeof(float),cudaMemcpyHostToDevice);
//     cudaMemcpy(M_d2,M_h2,n*p*sizeof(float),cudaMemcpyHostToDevice);
//     //cudaMemcpy(M_d3,M_h3,n*sizeof(float*),cudaMemcpyHostToDevice);

//     printf("Etape 3 \n");

//     MatrixMul<<<Nblock,Nthread>>>(M_d1,M_d2,M_d3,n);

//     cudaMemcpy(M_h3,M_d3,n*p*sizeof(float),cudaMemcpyDeviceToHost);

//     cudaFree(M_d1);
//     cudaFree(M_d2);
//     cudaFree(M_d3);

//     int s;
//     printf("Which value to disp? \n");
//     scanf("%d", &s);
//     printf("%f \n", M_h3[s]);
//     // for(int i=0;i<n;i++){
//     //     printf("Etape 3.1 \n");
//     //     cudaMalloc((void **) &(M_d1[i]), p*sizeof(float));
//     //     printf("Etape 3.2 \n");
//     //     cudaMemcpy(M_d1[i],M_h1[i],p*sizeof(float),cudaMemcpyHostToDevice);
//     //     cudaMalloc((void **) &(M_d2[i]), p*sizeof(float));
//     //     printf("Etape 3.2 \n");
//     //     cudaMemcpy(M_d2[i],M_h2[i],p*sizeof(float),cudaMemcpyHostToDevice);
//     // }


// }