
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__host__ void MatrixPrint(float* M, int n,int p){


    for(int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            if (i==0 && j==0){
                printf("[[ %f, " , M[0]);
            }

            else if(i==(n-1) && j==(p-1)){
                printf(" %f ]] \n" , M[n*i+j]);
            }

            else if (j==(p-1)){
               printf(" %f ]  \n" , M[n*i+j]); 
            }

            else if (j==0){
               printf("[ %f,  " , M[n*i+j]); 
            }

            else {
                printf(" %f, " , M[n*i+j]); 
            }
        }
    }


}

