#include "hip/hip_runtime.h"
#include "activation.cu"


__global__ void convolve2d(float* C, float* M, float* K, int n, int ksize){

    int idx= blockDim.x * blockIdx.x + threadIdx.x;
    int x= (idx%(n*n))/n;
    int y= idx%n; 
    int w = idx/(n*n);
    if (idx==0){
        printf("You are in the first convolve \n");
    }
    float tmp = float(0);
    
    for(int i=0; i<ksize;i++){
        for (int j=0; j<ksize;j++){
            if (idx==0){
                printf("%d \n", (x+i)*n+y+j);
                printf("K[(ksize-i)*ksize+(ksize-j)] = %f \n", K[(ksize-i-1)*ksize+(ksize-j-1)]);
                printf("M[(x+i)*n+y+j] = %f \n", M[(x+i)*n+y+j]);
                printf("K[(ksize-i)*ksize+(ksize-j)]*M[(x+i)*n+y+j] = %f \n", K[(ksize-i-1)*ksize+(ksize-j-1)]*M[(x+i)*n+y+j]);
                printf("tmp = %f \n", tmp);
            }
            tmp+=K[(ksize-i-1)*ksize+(ksize-j-1)+w*5*5]*M[(x+i)*n+y+j];
        }

    }
    
    C[idx]=activation(tmp/(n*n));
}


