#include "hip/hip_runtime.h"
#include <math.h>
#include "MatrixMul.cu"


__host__ int main(float* S,float theta){


    int n=512;
    float a = M_PI/180;

    int Nthread = 1024;
    int Nblock=(n*8+Nthread)/Nthread;
    
    float C[]={(float)1,(float)sqrt(2),(float)0,(float)sqrt(2),(float)0,(float)1,(float)1,(float)1,(float)0,(float)sqrt(2),(float)1,(float)0,(float) sqrt(2),(float) -sqrt(2),(float)0,(float)1, (float)-1,(float)sqrt(2),(float)0,(float)-sqrt(2),(float)1,(float)-sqrt(2),(float)0,(float)sqrt(2),(float)0,(float)1,(float)-1,(float)-1,(float)0,(float)sqrt(2),(float)1,(float)0,(float)-sqrt(2),(float)-sqrt(2),(float)0,(float)1,(float)1,(float)-1,(float)0,(float)-sqrt(2)}
;
    float Y[]={ 1.0, sqrt(2)*cos(a*theta),sqrt(2)*sin(a*theta),sqrt(2)*cos(2*a*theta),sqrt(2)*sin(2*a*theta)};
    float* Sn;

    float* C_d;
    float* Y_d;
    float* G_d;
    float* Sn_d;
    float* S_d;

    Sn=(float*)malloc(n*8*sizeof(float));

    hipMalloc((void **) &C_d,5*8*sizeof(float));
    hipMalloc((void **) &Y_d,5*sizeof(float));
    hipMalloc((void **) &G_d,8*sizeof(float));
    hipMalloc((void **) &Sn_d,n*8*sizeof(float));
    hipMalloc((void **) &S_d,8*sizeof(float));
 
    hipMemcpy(C_d,C,5*8*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(Y_d,Y,5*sizeof(float),hipMemcpyHostToDevice);

    MatrixMul<<<Nblock,Nthread>>>(C_d,Y_d,G_d,8,5,1);

    MatrixMul<<<Nblock,Nthread>>>(G_d,Sn_d,S_d,8,1,n);

    hipMemcpy(Sn,Sn_d,n*8*sizeof(float),hipMemcpyDeviceToHost);









}
