#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>




__global__ void MatrixAdd(float *Mat1,float *Mat2,float *Mat3, int n, int p){

    int idx= blockIdx.x* blockDim.x+ threadIdx.x;

    if (idx<n*p){
        Mat3[idx]=Mat2[idx]+Mat1[idx];
    }


}




__host__ int main (void){


    int n=5;
    int p=5;

    float *M_h1, *M_h2,*M_h3, *M_d1,*M_d2,*M_d3;
    printf("Etape 1 \n");
    M_h1=(float*)malloc(n*p*sizeof(float));
    M_h2=(float*)malloc(n*p*sizeof(float));
    M_h3=(float*)malloc(n*p*sizeof(float));
    // for (int m=0;m<n;m++){
    //     M_h1[m]=(float*)malloc(p*sizeof(float));
    //     M_h2[m]=(float*)malloc(p*sizeof(float));
    //     M_h3[m]=(float*)malloc(p*sizeof(float));
    // }

    int Nthread=p;
    int Nblock= n;
    
    for (int i=0;i<n*p;i++){
       
        M_h1[i]=(float)i;
        M_h2[i]=(float)i;
       
    }
    printf("Etape 2 \n");
    hipMalloc((void **) &M_d1, p*n*sizeof(float));
    hipMalloc((void **) &M_d2, p*n*sizeof(float));
    hipMalloc((void **) &M_d3, p*n*sizeof(float));
    hipMemcpy(M_d1,M_h1,n*p*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(M_d2,M_h1,n*p*sizeof(float),hipMemcpyHostToDevice);
    printf("Etape 3 \n");
    // for(int i=0;i<n;i++){
    //     printf("Etape 3.1 \n");
    //     cudaMalloc((void **) &(M_d1[i]), p*sizeof(float));
    //     printf("Etape 3.2 \n");
    //     cudaMemcpy(M_d1[i],M_h1[i],p*sizeof(float),cudaMemcpyHostToDevice);
    //     cudaMalloc((void **) &(M_d2[i]), p*sizeof(float));
    //     printf("Etape 3.2 \n");
    //     cudaMemcpy(M_d2[i],M_h2[i],p*sizeof(float),cudaMemcpyHostToDevice);
    // }

    MatrixAdd<<<Nblock,Nthread>>>(M_d1,M_d2,M_d3,n,p);

    hipMemcpy(M_h3,M_d3,n*p*sizeof(float),hipMemcpyDeviceToHost);

    // for(int i=0;i<n;i++){
    //     printf("Etape 3.1 \n");
    //     cudaMalloc((void **) &(M_d3[i]), p*sizeof(float));
    //     printf("Etape 3.2 \n");
    //     cudaMemcpy(M_d3[i],M_h3[i],p*sizeof(float),cudaMemcpyHostToDevice);

    // }




    // for(int i=0;i<n;i++){
    //     cudaFree(M_d1[i]);
    //     cudaFree(M_d2[i]);
    //     cudaFree(M_d3[i]);
    //     free(M_h1[i]);
    //     free(M_h2[i]);
    //     free(M_h3[i]);

    // }

    printf("%f \n", M_h3[5]);

    hipFree(M_d1);
    hipFree(M_d2);
    hipFree(M_d3);

    free(M_h1);
    free(M_h2);
    free(M_h3);

    return(0);

}