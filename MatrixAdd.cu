#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>




__global__ void MatrixAdd(float *Mat1,float *Mat2,float *Mat3, int n, int p){

    int idx= blockIdx.x* blockDim.x+ threadIdx.x;

    if (idx<n*p){
        Mat3[idx]=Mat2[idx]+Mat1[idx];
    }


}

